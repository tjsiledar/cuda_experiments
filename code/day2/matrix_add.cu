#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 100

__global__ void kernelMatrixAddElementWise(float *A, float *B, float *C){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx<N*N) C[idx] = A[idx] + B[idx];
}


void matrixAdd(float A[N][N], float B[N][N], float C[N][N]){
    // initialize device memory pointers
    float *d_A, *d_B, *d_C;
    int size = N*N*sizeof(float);

    // allocate memory in device memory
    hipMalloc((void **)&d_A, size); 
    hipMalloc((void **)&d_B, size); 
    hipMalloc((void **)&d_C, size); 

    // copy the matrices from the host to device memory
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
    
    // call the kernel function to perform matrix addition
    kernelMatrixAddElementWise<<<100, 100>>>(d_A,d_B,d_C);

    // copy back the result matrix from device to host memory
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
    
    // free cuda memory
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
}


int main(){
    float A[N][N], B[N][N], C[N][N];
    
    // initialize the matrices
    for(int i=0; i<N; i++){
        for(int j=0; j<N; j++){
            A[i][j] = (i+j)*0.1f;
            B[i][j] = (i+j)*0.2f;
        }
    }

    matrixAdd(A, B, C);
    
    // print the first few elements
    for(int i=0; i<5; i++){
        for(int j=0; j<5; j++){
            printf("%f, %f, %f\t", A[i][j], B[i][j], C[i][j]);
        }
        printf("\n");
    }
    
    return 0;
}