#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void vecAddKernel(float *A, float *B, float *C, int N){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    C[i] = A[i] + B[i];
}

int main(){
    // define the variables
    // host variables
    int N=10000;
    int size = N*sizeof(float);
    float A[N], B[N], C[N];

    // device variables
    float *d_A, *d_B, *d_C;

    // loop over the arrays in the host memory
    for(int i=0; i<10000; i++){
        A[i] = i*1.1f;
        B[i] = i*1.2f;
    }

    // allocate memory in the device memory and use device pointers to assign them
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    // copy the arrays from the host memory to the device memory
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    // call the kernel func to compute the vector addition
    vecAddKernel<<<ceil(N/1000.0), 1000>>>(d_A, d_B, d_C, N);
    
    // copy the array from device back to the host memory
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
    
    // print the arrays
    for(int i=0; i<5; i++){
        printf("%f, %f, %f\n", A[i], B[i], C[i]);
    }

    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    
    return 0;
}